#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <chrono>

int main() {
    int M, N, K;
    M = N = K = 512;
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    srand(time(0));
    for (int i = 0; i < M * K; ++i) {
        h_A[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
    for (int i = 0; i < K * N; ++i) {
        h_B[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, M * K * sizeof(float));
    hipMalloc((void **)&d_B, K * N * sizeof(float));
    hipMalloc((void **)&d_C, M * N * sizeof(float));
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f;
    float beta = 0.0f;
    auto start = std::chrono::high_resolution_clock::now();
    // 执行矩阵乘法
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Run time: " << elapsed.count() << " s" << std::endl;
    std::cout << "Matrix sizes: "<< M << std::endl;
    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}